
#include <hip/hip_runtime.h>
__global__ void EPRI_3D_kernel(double *object,double *projection,double *GX, double *GY,double *GZ,double length_of_finalcube,double number_of_finalimage,double length_of_projection,double number_of_angle,double number_of_projection,double ii,double interp_method)
{


double xx,yy,zz,t,t0,value1,value2;
int kk,m,n,k,t1,t2;
ii=(int)ii;

 m=threadIdx.x;
 n=blockIdx.x;
 k=ii-1;

 xx=(m+1-number_of_finalimage/2)*length_of_finalcube/number_of_finalimage;
 yy=(n+1-number_of_finalimage/2)*length_of_finalcube/number_of_finalimage;
 zz=(k+1-number_of_finalimage/2)*length_of_finalcube/number_of_finalimage;
                
                


for(kk=1;kk<=number_of_angle;kk++)
{
 if(int(interp_method)==0)
        {
         t=xx*GX[kk-1]+yy*GY[kk-1]+zz*GZ[kk-1];
         t=round(t/(length_of_projection/number_of_projection));
         t=t+(number_of_projection/2);
         if(t>=0&&t<=(number_of_projection-1))
         object[(int)(n*number_of_finalimage+m)]+=projection[(int)((kk-1)*number_of_projection+t)];
        }
        else
        {
         t=xx*GX[kk-1]+yy*GY[kk-1]+zz*GZ[kk-1];
         t0=t/(length_of_projection/number_of_projection);
         t1=floor(t/(length_of_projection/number_of_projection));
         t2=ceil(t/(length_of_projection/number_of_projection));
         t0=t0+(number_of_projection/2);
         t1=t1+(number_of_projection/2);
         t2=t2+(number_of_projection/2);

         if(t1>=0&&t2<=(number_of_projection-1))
             {
              value1=projection[(int)((kk-1)*number_of_projection+t1)];
              value2=projection[(int)((kk-1)*number_of_projection+t2)];
              object[(int)(n*number_of_finalimage+m)]+=value1+(value2-value1)*(t0-t1);
             }
        }
}

}